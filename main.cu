#include "hip/hip_runtime.h"
//nvcc -Xcompiler -fopenmp  main.cu -o main && ./main Examples/t.txt Examples/test_ME.txt Examples/test_CC1.txt Examples/test_map.txt Examples/test_change1.txt 2 1

#include <iostream>
#include <fstream>
#include <vector>
#include <stdio.h>
#include <string.h>
#include <map>
#include <cmath>
#include <omp.h>
#include <unordered_map>
#include "DataStructure.hpp"
#include "ReadData.hpp"
#include "CreateGraph.hpp"
//#include "TraverseMeta.hpp"
#include "PrintFunctions.hpp"

using namespace std;

typedef pair<int,int> int_int;

// #define DEBUG 

// CUDA kernel to mark hubs
__global__ void markHubs(int_int* inserts_meta, MetaNode* MN_list, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int src = inserts_meta[i].first;
        int dest = inserts_meta[i].second;
        MN_list[src].is_hub = true;
        MN_list[dest].is_hub = true;
    }
}

// CUDA kernel to process nodes
__global__ void processNodes(int* g_meta_out_deg, int* g_meta_in_deg, int* gI_meta_out_deg, int* gI_meta_in_deg,
                             MetaNode* MN_list, int* Hub_Id, int N, int* non_zero, int trim_th, int hub_th_high, int hub_th_low) {
    int mn = blockIdx.x * blockDim.x + threadIdx.x;
    if (mn < N) {
        if (g_meta_out_deg[mn] <= trim_th && gI_meta_out_deg[mn] <= trim_th) {
            MN_list[mn].trimmed = true;
            MN_list[mn].is_hub = false;
            return;
        }
        if (g_meta_in_deg[mn] <= trim_th && gI_meta_in_deg[mn] <= trim_th) {
            MN_list[mn].trimmed = true;
            MN_list[mn].is_hub = false;
            return;
        }

        if (gI_meta_out_deg[mn] > 0)
            MN_list[mn].down = 1;
        if (gI_meta_in_deg[mn] > 0)
            MN_list[mn].up = 1;

        if ((g_meta_out_deg[mn] > hub_th_high || g_meta_in_deg[mn] > hub_th_high) &&
            (g_meta_out_deg[mn] > hub_th_low && g_meta_in_deg[mn] > hub_th_low)) {
            MN_list[mn].Hub_info[*non_zero] = 5;
            MN_list[mn].h_idx = *non_zero;
            Hub_Id[*non_zero] = mn;

            MN_list[mn].up = 1;
            MN_list[mn].down = 1;

            

            atomicAdd(non_zero, 1);
            return;
        }

        if (MN_list[mn].is_hub == true) {
            MN_list[mn].Hub_info[*non_zero] = 5;
            MN_list[mn].h_idx = *non_zero;
            MN_list[mn].up = 1;
            MN_list[mn].down = 1;

            

            atomicAdd(non_zero, 1);
        }
    }
}
int main(int argc, char *argv[]) {
    

    check_file_open(argv[1], argv[2], argv[3], argv[4], argv[5], argc);
    int n, N, m, M; // number of nodes and metanodes
    int p = atoi(argv[7]); // number of threads

    double start_timer, st;

    int *e_src, *e_dest, *e_wt, *m_src, *m_dest, *m_wt;
    bool* insert_status;
    bool* delete_status;

    Graph g, g_meta, gI_meta;

    vector<int> SCCx; // SCC IDs for graph nodes
    unordered_map<int, int> sccMAP; // Mapping of SCCID from old to new 0 to N Continuous
    vector<int_int> edge_list, inserts, deletes, inserts_meta, deletes_meta;
    inserts.clear();
    deletes.clear();
    inserts_meta.clear();
    deletes_meta.clear();
    

    printf("Threads: %d \n", p);

    // ******************* READING FILES ****************************
    start_timer = omp_get_wtime();
    read_graph(argv[1], n, m, e_src, e_dest, e_wt); // reading the graph data
    read_graph(argv[2], N, M, m_src, m_dest, m_wt); // reading the metagraph data
    read_scc(argv[3], n, &SCCx);
    read_sccmap(argv[4], &sccMAP);
    read_changes(argv[5], &inserts, &deletes, &inserts_meta, &deletes_meta, &SCCx, &sccMAP);
    color("purple");
    printf("\n Time for Reading: %f \n", (float)(omp_get_wtime() - start_timer));
    color("reset");

    
    // ******************* READING COMPLETED ****************************

    // Processing Inserts
    st = omp_get_wtime();
    sort(inserts_meta.begin(), inserts_meta.end(), sort_first_second);
    inserts_meta.erase(unique(inserts_meta.begin(), inserts_meta.end()), inserts_meta.end());
    int insert_percent = floor((double)inserts_meta.size() / (double)M * 100.0);

    int *ins_src = new int[inserts_meta.size()];
    int *ins_dest = new int[inserts_meta.size()];
    for (int i = 0; i < inserts_meta.size(); i++) {
        ins_src[i] = inserts_meta[i].first;
        ins_dest[i] = inserts_meta[i].second;
    }

    printf("%d  %d %d", M, inserts_meta.size(), insert_percent);
    color("purple");
    printf("\n Time for Processing Inserts: %f \n", (float)(omp_get_wtime() - st));
    color("reset");

    // ******************* CREATING GRAPHS ****************************
    st = omp_get_wtime();
    create_graph(e_src, e_dest, e_wt, n, m, &g); // GRAPH
    create_graph(m_src, m_dest, m_wt, N, M, &g_meta); // METAGRAPH
    create_graph(ins_src, ins_dest, e_wt, N, inserts_meta.size(), &gI_meta); // INSERTION GRAPH
    color("purple");
    printf("\n Time for Creating graph: %f \n", (float)(omp_get_wtime() - st));
    color("reset");
    // ******************* CREATING GRAPHS COMPLETED ****************************

    st = omp_get_wtime();
    MetaNode* MN_list = new MetaNode[N]; // Allocate MN_list on host
    MetaNode* d_MN_list;
    int_int* d_inserts_meta;
    int* d_Hub_Id;
    int* d_non_zero;
    int* d_g_meta_out_deg;
    int* d_g_meta_in_deg;
    int* d_gI_meta_out_deg;
    int* d_gI_meta_in_deg;

    int hubX = 0;
    int non_zero = 0;
    int trim_th = 0; // nodes with degree <= threshold are trimmed
    int hub_th_high = 30; // nodes with degree >= threshold are hubs
    int hub_th_low = 3; // nodes with degree >= threshold are hubs

    // Allocate device memory
    hipMalloc(&d_inserts_meta, inserts_meta.size() * sizeof(int_int));
    hipMalloc(&d_MN_list, N * sizeof(MetaNode));
    hipMalloc(&d_Hub_Id, 1000 * sizeof(int));
    hipMalloc(&d_non_zero, sizeof(int));
    hipMalloc(&d_g_meta_out_deg, N * sizeof(int));
    hipMalloc(&d_g_meta_in_deg, N * sizeof(int));
    hipMalloc(&d_gI_meta_out_deg, N * sizeof(int));
    hipMalloc(&d_gI_meta_in_deg, N * sizeof(int));

    // Copy data to device
    hipMemcpy(d_inserts_meta, inserts_meta.data(), inserts_meta.size() * sizeof(int_int), hipMemcpyHostToDevice);
    hipMemcpy(d_MN_list, MN_list, N * sizeof(MetaNode), hipMemcpyHostToDevice);
    hipMemcpy(d_non_zero, &non_zero, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_g_meta_out_deg, g_meta.out_deg, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_g_meta_in_deg, g_meta.in_deg, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_gI_meta_out_deg, gI_meta.out_deg, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_gI_meta_in_deg, gI_meta.in_deg, N * sizeof(int), hipMemcpyHostToDevice);

    // Define block and grid sizes
    int blockSize = 256;
    int numBlocks = (inserts_meta.size() + blockSize - 1) / blockSize;

    // Launch kernel to mark hubs
    markHubs<<<numBlocks, blockSize>>>(d_inserts_meta, d_MN_list, inserts_meta.size());
    hipDeviceSynchronize();

    hipMemcpy(MN_list, d_MN_list, N * sizeof(MetaNode), hipMemcpyDeviceToHost);
    // Count the number of hubs
    for (int mn = 0; mn < N; mn++) {
        if (MN_list[mn].is_hub == true)
            hubX++;
    }
    printf("HUBs %d", hubX);

    numBlocks = (N + blockSize - 1) / blockSize;

    // Launch kernel to process nodes
    processNodes<<<numBlocks, blockSize>>>(d_g_meta_out_deg, d_g_meta_in_deg, d_gI_meta_out_deg, d_gI_meta_in_deg,
                                           d_MN_list, d_Hub_Id, N, d_non_zero, trim_th, hub_th_high, hub_th_low);
    hipDeviceSynchronize();

    // Copy results back to host
    hipMemcpy(MN_list, d_MN_list, N * sizeof(MetaNode), hipMemcpyDeviceToHost);
    hipMemcpy(&non_zero, d_non_zero, sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_inserts_meta);
    hipFree(d_MN_list);
    hipFree(d_Hub_Id);
    hipFree(d_non_zero);
    hipFree(d_g_meta_out_deg);
    hipFree(d_g_meta_in_deg);
    hipFree(d_gI_meta_out_deg);
    hipFree(d_gI_meta_in_deg);


    

    color("purple");
    printf("\n Time for Finding Hubs: %f \n", (float)(omp_get_wtime() - st));
    color("reset");
    non_zero++;
    printf("\n N: %d Hubs: %d \n", N, non_zero);

    delete[] MN_list; // Free host memory
    delete[] ins_src;
    delete[] ins_dest;

    return 0;
}
